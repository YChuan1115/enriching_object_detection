#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
// #include "common/hip/hip_runtime_api.h"
#include "cudaConvFFTData.h"
#include "cudaConvFFTData.cuh"

static bool debug = false;

enum OUT_INDEX{
    CONVOLUTION_CELL_INDEX
};

enum IN_INDEX{
    DATA_INDEX,
    MAX_KERNEL_H_INDEX,
    MAX_KERNEL_W_INDEX,
    KERNLE_CELL_INDEX,
    THREAD_SIZE_INDEX, // Optional
    GPU_INDEX          // Optional
};

////////////////////////////////////////////////////////////////////////////////
// Mex Entry
////////////////////////////////////////////////////////////////////////////////
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    char const * const errId = "cudaConvFFTData:InvalidInput";

    /* Choose a reasonably sized number of threads for the block. */
    int THREAD_PER_BLOCK_H = 16;
    int THREAD_PER_BLOCK_W = 8;
    int THREAD_PER_BLOCK_D = 8;
    int THREAD_PER_BLOCK_2D = 32;

    /* Initialize the MathWorks GPU API. */
    // If initialized mxInitGPU do nothing
    if (mxInitGPU() != MX_GPU_SUCCESS)
        mexErrMsgTxt("mxInitGPU fail");
    

    /* Throw an error if the number of inputs mismatch */
    if ( (nrhs <  (KERNLE_CELL_INDEX + 1)) || (nrhs > (GPU_INDEX + 1) ))
        mexErrMsgIdAndTxt(errId, "Wrong number of inputs");


    /*  Set data */
    const mxArray *mxDATA = prhs[DATA_INDEX];
    if (mxIsGPUArray(mxDATA) || 
            mxGetNumberOfDimensions(mxDATA) != 3 || 
            mxGetClassID(mxDATA) != mxSINGLE_CLASS)
        mexErrMsgTxt("Invalid data input");


    /* Kernel dimensions */
    int MAX_KERNEL_H = (int)mxGetScalar(prhs[MAX_KERNEL_H_INDEX]);
    int MAX_KERNEL_W = (int)mxGetScalar(prhs[MAX_KERNEL_W_INDEX]);
    if(debug) fprintf(stderr,"Kernel size: h=%d, w=%d\n",MAX_KERNEL_H,MAX_KERNEL_W);


    /* Kernel Input */
    if (mxGetClassID(prhs[KERNLE_CELL_INDEX]) != mxCELL_CLASS)
        mexErrMsgIdAndTxt(errId, "Kernel must be a cell array");
    mwSize nKernel = mxGetNumberOfElements(prhs[KERNLE_CELL_INDEX]);
    int N_KERNEL = (int)nKernel;
    if(debug) fprintf(stderr,"N Kernel: %d\n", N_KERNEL);


    /* Thread size */
    if (( nrhs > THREAD_SIZE_INDEX)  && mxGetNumberOfElements(prhs[THREAD_SIZE_INDEX]) != 4)
        mexErrMsgIdAndTxt(errId, "CUDA Thread Size must be 4 integers : THREAD_PER_BLOCK_H, THREAD_PER_BLOCK_W, THREAD_PER_BLOCK_D, THREAD_PER_BLOCK_2D\nYou must choose size such that total thread will not be larger than MaxThreadsPerBlock");

    if ( nrhs > THREAD_SIZE_INDEX ){
        const double* threadSize = (double *)mxGetData(prhs[THREAD_SIZE_INDEX]);
        THREAD_PER_BLOCK_H = (int)threadSize[0];
        THREAD_PER_BLOCK_W = (int)threadSize[1];
        THREAD_PER_BLOCK_D = (int)threadSize[2];
        THREAD_PER_BLOCK_2D = (int)threadSize[3];
        if(debug) fprintf(stderr,"Thread size: H=%d, W=%d, D=%d, 2D=%d\n", THREAD_PER_BLOCK_H, THREAD_PER_BLOCK_W, THREAD_PER_BLOCK_D, THREAD_PER_BLOCK_2D);
    }

    int GPU_ID = 0;
    if (nrhs > GPU_INDEX ){
       GPU_ID = (int)mxGetScalar(prhs[GPU_INDEX]); 
       if(debug) fprintf(stderr,"Using GPU : %d\n", GPU_ID);
       hipSetDevice(GPU_ID);
    }


    /*  FFT Data */
    // Data dimensions
    const mwSize *DATA_dims = mxGetDimensions(mxDATA);
    int DATA_H = DATA_dims[0];
    int DATA_W = DATA_dims[1];
    int FEATURE_DIM = DATA_dims[2];

    float *h_Data = (float *)mxGetData(mxDATA);
    if(debug) fprintf(stderr,"Data size: h=%d, w=%d, f=%d\n",DATA_H,DATA_W,FEATURE_DIM); 

    // Width and height of padding
    int PADDING_H = MAX_KERNEL_H - 1;
    int PADDING_W = MAX_KERNEL_W - 1;

    // Derive FFT size from data and kernel dimensions
    // FFT_H = computeFFTsize(DATA_H + PADDING_H);
    // FFT_W = computeFFTsize(DATA_W + PADDING_W);
    int FFT_H = computeFFTsize16(DATA_H + PADDING_H);
    int FFT_W = computeFFTsize16(DATA_W + PADDING_W);
    int CFFT_W = FFT_W;
    int CFFT_H = FFT_H/2 + 1;

    if(debug) fprintf(stderr,"FFT size: h=%d, w=%d\n",FFT_H,FFT_W);

    int DATA_SIZE = DATA_W * DATA_H * FEATURE_DIM * sizeof(float);
    int FFT_SIZE  = FFT_W  * FFT_H  * FEATURE_DIM * sizeof(float);
    int CFFT_SIZE = CFFT_W * CFFT_H * FEATURE_DIM * sizeof(float2);
    int CONV_SIZE = FFT_W  * FFT_H  * sizeof(float);
    
    int BATCH = FEATURE_DIM;
    int FFT_Dims[] = { FFT_W, FFT_H };
    int CFFT_Dims[] = { CFFT_W, CFFT_H };
    int idist = FFT_W * FFT_H;
    int odist = CFFT_W * CFFT_H;

    hipfftHandle FFTplan_R2C, FFTplan_C2R;
    CUFFT_SAFE_CALL(hipfftPlanMany(&FFTplan_R2C, 
        2, // rank
        FFT_Dims, 
        FFT_Dims, 1, idist, // *inembed, istride, idist
        CFFT_Dims, 1, odist, // *onembed, ostride, odist
        HIPFFT_R2C, 
        BATCH)); // batch

    CUFFT_SAFE_CALL(hipfftPlanMany(&FFTplan_C2R, 
        2, // rank
        FFT_Dims,
        CFFT_Dims, 1, odist, // *inembed, istride, idist
        FFT_Dims, 1, idist, // *onembed, ostride, odist
        HIPFFT_C2R, 
        BATCH)); // batch

    float *d_Data;
    float *d_PaddedData;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_Data,         DATA_SIZE));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_PaddedData,   FFT_SIZE));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_Data, h_Data, DATA_SIZE, hipMemcpyHostToDevice));

    dim3 threadBlock3D(THREAD_PER_BLOCK_H, THREAD_PER_BLOCK_W, THREAD_PER_BLOCK_D);
    dim3 dataBlockGrid3D( iDivUp(FFT_W, threadBlock3D.x), 
                        iDivUp(FFT_H, threadBlock3D.y), 
                        iDivUp(FEATURE_DIM, threadBlock3D.z));

    padData<<<dataBlockGrid3D, threadBlock3D>>>(
        d_PaddedData,
        d_Data,
        FFT_W,
        FFT_H,
        DATA_W,
        DATA_H,
        FEATURE_DIM
        );

    hipfftComplex *d_CFFT_DATA;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_CFFT_DATA,     CFFT_SIZE));
    CUFFT_SAFE_CALL(hipfftExecR2C(FFTplan_R2C, d_PaddedData, d_CFFT_DATA));
    CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());
    hipFree(d_Data);




    

    /* Convolution FFT */
    // Set Variables 
    float *d_IFFTEProd;
    float *d_CONVOLUTION;
    hipfftComplex *d_CFFT_KERNEL;
    hipfftComplex *d_FFTEProd;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_IFFTEProd,    FFT_SIZE));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_CONVOLUTION,  CONV_SIZE));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_CFFT_KERNEL,  CFFT_SIZE));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_FFTEProd,     CFFT_SIZE));
    
    const mxArray *mxCurrentCell;
    const mxGPUArray *mxKernel;
    const mwSize *mxKernel_Dim;
    float *h_Kernel;
    float *d_Kernel;
    int KERNEL_H, KERNEL_W, KERNEL_SIZE;

    dim3 threadBlock2D( THREAD_PER_BLOCK_2D, THREAD_PER_BLOCK_2D);
    dim3 dataBlockGrid2D( iDivUp(FFT_W, threadBlock2D.x), 
                        iDivUp(FFT_H, threadBlock2D.y));
    
    mwSize mwCONV_Dims[2];
    mwCONV_Dims[0] = FFT_H;
    mwCONV_Dims[1] = FFT_W;

    plhs[CONVOLUTION_CELL_INDEX] = mxCreateCellMatrix(1, N_KERNEL);

    for (int kernelIdx = 0; kernelIdx < N_KERNEL; kernelIdx++){
        
        // Get Kernel Data
        mxCurrentCell = mxGetCell(prhs[KERNLE_CELL_INDEX], kernelIdx);
        if (!mxIsGPUArray(mxCurrentCell)){
            
            if( mxGetClassID(mxCurrentCell) != mxSINGLE_CLASS || mxGetNumberOfDimensions(mxCurrentCell) != 3 )
                mexErrMsgIdAndTxt(errId, "Kernels must be of type float and have features larger than 1");

            h_Kernel = (float *)mxGetData(mxCurrentCell);
            mxKernel_Dim = mxGetDimensions(mxCurrentCell);

            // Kernel dimensions
            KERNEL_H = mxKernel_Dim[0];
            KERNEL_W = mxKernel_Dim[1];
            KERNEL_SIZE = KERNEL_W * KERNEL_H * FEATURE_DIM * sizeof(float);

            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_Kernel, KERNEL_SIZE));
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_Kernel, h_Kernel, KERNEL_SIZE, hipMemcpyHostToDevice));
            mxKernel = NULL;
        }else{ // Kernel is GPU Array
            mxKernel = mxGPUCreateFromMxArray(mxCurrentCell);

            if ( mxGPUGetClassID(mxKernel) != mxSINGLE_CLASS || mxGPUGetNumberOfDimensions(mxKernel) != 3 )
                mexErrMsgIdAndTxt(errId, "Kernels must be of type float and have features larger than 1");

            mxKernel_Dim = mxGPUGetDimensions(mxKernel);

            // Kernel dimensions
            KERNEL_H = mxKernel_Dim[0];
            KERNEL_W = mxKernel_Dim[1];
            KERNEL_SIZE = KERNEL_W * KERNEL_H * FEATURE_DIM * sizeof(float);

            d_Kernel = (float *)mxGPUGetDataReadOnly(mxKernel);
        }

        if(debug) fprintf(stderr,"Kernel size: h=%d, w=%d\n", KERNEL_H, KERNEL_W);

        if (FEATURE_DIM != mxKernel_Dim[2] || KERNEL_W > FFT_W || KERNEL_H > FFT_H )
            mexErrMsgIdAndTxt(errId, "Kernel and Data must have the same number of features and kernel size should be smaller than data size");

        padData<<<dataBlockGrid3D, threadBlock3D>>>(
                d_PaddedData,
                d_Kernel,
                FFT_W,
                FFT_H,
                KERNEL_W,
                KERNEL_H,
                FEATURE_DIM
            );

        CUFFT_SAFE_CALL(hipfftExecR2C(FFTplan_R2C, d_PaddedData, d_CFFT_KERNEL));
        CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

        if(debug) fprintf(stderr,"FFT done\n");

        
        /* Hadamard product, Element-wise multiplication in frequency domain */
        /* If execute the following, second compile of this file create MATLAB error */
        elementwiseProductAndNormalize<<<dataBlockGrid3D, threadBlock3D>>>(
                d_FFTEProd, // out
                d_CFFT_DATA, // in data
                d_CFFT_KERNEL,   // in kernel
                CFFT_H,
                CFFT_W,
                FEATURE_DIM,
                1.0f / (FFT_W * FFT_H)
            );

        CUFFT_SAFE_CALL(hipfftExecC2R(FFTplan_C2R, d_FFTEProd, d_IFFTEProd));
        CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

        sumAlongFeatures<<<dataBlockGrid2D, threadBlock2D>>>(
                d_CONVOLUTION,
                d_IFFTEProd,
                FFT_H,
                FFT_W,
                FEATURE_DIM
            );

        mxArray * convolutionResult = mxCreateNumericArray(2, mwCONV_Dims, mxSINGLE_CLASS, mxREAL);
        float * h_CONVOLUTION = (float *)mxGetData(convolutionResult);
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(h_CONVOLUTION, d_CONVOLUTION, CONV_SIZE ,hipMemcpyDeviceToHost));

        mxSetCell(plhs[CONVOLUTION_CELL_INDEX], kernelIdx, convolutionResult);
        if(mxKernel == NULL) hipFree(d_Kernel);
        else mxGPUDestroyGPUArray(mxKernel);
    }
    // plhs[1] = mxGPUCreateMxArrayOnGPU(mxFFTKernel);

    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    // mxGPUDestroyGPUArray(mxFFTData);
    // mxGPUDestroyGPUArray(mxConvolution);    
    // mxGPUDestroyGPUArray(mxFFTKernel);
    
    hipfftDestroy(FFTplan_R2C);
    hipfftDestroy(FFTplan_C2R);

    hipFree(d_CFFT_DATA);
    hipFree(d_IFFTEProd);
    hipFree(d_CONVOLUTION);
    hipFree(d_CFFT_KERNEL);
    hipFree(d_FFTEProd);
    hipFree(d_PaddedData);
}
